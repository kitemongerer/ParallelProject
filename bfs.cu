#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <vector>

// CUDA runtime
#include <hip/hip_runtime.h>

using namespace std;

// Thread block size
#define TBS 512

// Warp size
#define WS 32

class Node {
private:
	int value;
	int* children;
	int numChildren;
	int explored;

public:
	Node();
	Node(int);
	__host__ __device__ int getValue();
	void addChild(Node*);
	__host__ __device__ int* getChildren();
	__host__ __device__ int getNumChildren();
	void printNode();
	void initializeChildren(int);
	__host__ __device__ int getExplored();
	void setExplored(int);
	__device__ int parallelSetExplored(int);
};

__global__ void childListExploreWave(int *d_waveMask, int *d_nextWaveMask, int *d_children, int *d_numChildren, int *d_cost, int *d_size, int *d_maxChildren) {
	int idx = blockIdx.x * TBS + threadIdx.x;

	if (idx < *d_size && d_waveMask[idx] == 1) {
		int numChildren = d_numChildren[idx];
		
		for (int i = 0; i < numChildren; i++) {
			int child = d_children[idx * *d_maxChildren + i];
			
			atomicCAS(&d_nextWaveMask[child],0,1);
					
			if (d_waveMask[child] == 0) {
				d_cost[child] = d_cost[idx] + 1;
			}
		}
	}
	if(idx < *d_size && d_waveMask[idx] == 2){
		d_nextWaveMask[idx] = 2;
	}
}

__global__ void exploreWave(int *d_waveMask, int *d_nextWaveMask, Node *d_graph, int *d_children, int *d_cost, int *d_size, int *d_maxChildren) {
	int idx = blockIdx.x * TBS + threadIdx.x;

	if (idx < *d_size && d_waveMask[idx] == 1) {

		//printf("%i child\n", d_children[idx]);
		Node currentNode = d_graph[idx];
		int numChildren = currentNode.getNumChildren();
		
		for (int i = 0; i < numChildren; i++) {
			int child = d_children[idx * *d_maxChildren + i];
			
			atomicCAS(&d_nextWaveMask[child],0,1);
					
			if (d_waveMask[child] == 0) {
				//printf("%i child: %i\n\n\n", idx, child);
				d_cost[child] = d_cost[idx] + 1;
				//d_graph[children[i]].parallelSetExplored(1);	
			}
		}
	}
	if(idx < *d_size && d_waveMask[idx] == 2){
		d_nextWaveMask[idx] = 2;
	}
}

__global__ void setPreviousExplored(int *d_waveMask, int *d_nextWaveMask, int *d_size){
	int idx = blockIdx.x * TBS + threadIdx.x;

	if(idx < *d_size){
		if(d_waveMask[idx] == 1){
			d_nextWaveMask[idx] = 2;
		}
	}
}

int* generateChildren(Node *nodes, int nNodes, int maxEdgesPerNode) {
	int* children = new int[nNodes * maxEdgesPerNode];

	for (int i = 0; i < nNodes; i++) {
		int numEdges = (rand() % maxEdgesPerNode) + 1;
		nodes[i].initializeChildren(numEdges);
		for (int j = 0; j < numEdges; j++) {
			int child = rand() % nNodes;
			bool isChild = false;
			for (int k = 0; k < nodes[i].getNumChildren(); k++){
				if (child == nodes[i].getChildren()[k]){
					isChild = true;
					break;
				}
			}
			if (!isChild && child != nodes[i].getValue()){
				children[i * maxEdgesPerNode + nodes[i].getNumChildren()] = child;
				nodes[i].addChild(&nodes[child]);
			}
		}
	}
	
	for (int i = 0; i < nNodes; i++) {
		nodes[i].printNode();
	}

	return children;
}

Node* generateGraph(int nNodes) {
	srand((unsigned)time(0)); 
	Node* nodes = new Node[nNodes];
	
	for (int i = 0; i < nNodes; i++) {
		Node* tmp = new Node(i);
		nodes[i] = *tmp;
	}

	return nodes; 
}

void exploreChild(Node* child, vector< vector<Node*> >* path, int depth, Node* nodes) {
	//printf("Explore Child%i Depth: %i\n", child->getValue(), depth);
	
	if (child->getNumChildren() > 0) {
		vector<Node*> newPath;
		if (path->size() <= depth) {
			path->push_back(newPath);
		}
		vector<Node*>* currentPath = &(path->at(depth));
		//printf("%i numChildren: %i\n", child->getValue(), child->getNumChildren());
		//child->printNode();
		for (int i = 0; i < child->getNumChildren(); i++) {
			Node* newChild = &nodes[child->getChildren()[i]];
			if (newChild->getExplored() == 0) {
				currentPath->push_back(newChild);
				newChild->setExplored(1);
			}
		}

		// Explore loop after push loop so it is actually BFS
		for (int i = 0; i < child->getNumChildren(); i++) {
			Node* newChild = &nodes[child->getChildren()[i]];
			if (newChild->getExplored() == 0) {
				exploreChild(newChild, path, depth + 1, nodes);	
			}
		}
	}

	child->setExplored(2);
	return;
}

vector< vector<Node*> > bfs(Node* nodes, int size) {
	vector< vector<Node*> > path;

	Node* currentNode = &nodes[0];
	vector<Node*> firstPath;
	firstPath.push_back(currentNode);
	path.push_back(firstPath);

	currentNode->setExplored(1);
	exploreChild(currentNode, &path, 1, nodes);

	return path;
}

int* transformBfs(vector< vector<Node*> > path, int size) {
	int *result = new int[size];
	for (int i = 0; i < path.size(); i++) {
		printf("%i - ", i);
		for (int j = 0; j < path[i].size(); j++) {
			printf(" %i ", path[i][j]->getValue());
			result[path[i][j]->getValue()] = i;
		}
		printf("\n");
	}
	return result;
}

int* transformNumChildren(Node* nodes, int size) {
	int *result = new int[size];
	for (int i = 0; i < size; i++) {
		result[i] = nodes[i].getNumChildren();
	}
	return result;
}

void callChildListExploreWave(int *d_size, int *d_children, int *d_numChildren, int size, int *d_maxChildren, int *synchResult) {
	hipEvent_t start;
	hipEventCreate(&start);
    hipEvent_t stop;
    hipEventCreate(&stop);

    int *d_cost, *d_waveMask, *d_nextWaveMask;

	// Allocate space for device copies
	hipMalloc((void **)&d_cost, size * sizeof(int));
	hipMalloc((void **)&d_waveMask, size * sizeof(int));
	hipMalloc((void **)&d_nextWaveMask, size * sizeof(int));


    int gridSz = ceil(((float) size) / TBS);
    // Record the start event
    hipEventRecord(start, NULL);

    int *waveMask = new int[size];
    int *nextWaveMask = new int[size]; 

    int *cost = new int[size];
    cost[0] = 0;
    for (int i = 1; i < size; i++) {
    	cost[i] = -1;
    	waveMask[i] = 0;
		nextWaveMask[i] = 0;
    }

    waveMask[0] = 1;

    hipMemcpy(d_cost, cost, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_waveMask, waveMask, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_nextWaveMask, nextWaveMask, size * sizeof(int), hipMemcpyHostToDevice);
    
    bool complete = false;
    while(!complete) {

    	// Launch kernel on GPU
		childListExploreWave<<<gridSz, TBS>>>(d_waveMask, d_nextWaveMask, d_children, d_numChildren, d_cost, d_size, d_maxChildren);
		hipDeviceSynchronize();
		setPreviousExplored<<<gridSz, TBS>>>(d_waveMask, d_nextWaveMask, d_size);		
		hipDeviceSynchronize();
		hipMemcpy(d_waveMask, d_nextWaveMask, size * sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(d_nextWaveMask, nextWaveMask, size * sizeof(int), hipMemcpyHostToDevice);

		complete = true;
		hipMemcpy(waveMask, d_waveMask, size * sizeof(int), hipMemcpyDeviceToHost);
		for(int i = 0 ; i < size; i++){
			if(waveMask[i] == 1){
				complete = false;
			}
		}
		printf("\n");
    }

	
	
	// Make sure result is finished
	hipDeviceSynchronize();

	// Record end event
	hipEventRecord(stop, NULL);
	hipEventSynchronize(stop);
	float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);

    printf("GPU Time= %.3f msec\n", msecTotal);

	// Copy result back to host
	int *gpu_result = (int *) malloc(size * sizeof(int));
	hipMemcpy(gpu_result, d_cost, size * sizeof(int), hipMemcpyDeviceToHost);

	bool isCorrect = true;

	for (int j = 0; j < size; j++) {
		printf("%i cost: %i\n", j, gpu_result[j]);
	}


	for (int i = 0; i < size; i++) {
		if (synchResult[i] != gpu_result[i]) {
			isCorrect = false;
			printf("%i CPU: %i GPU:%i\n", i, synchResult[i], gpu_result[i]);
		}
	}

	if (!isCorrect) {
		printf("The results do not match\n");
	} else {
		printf("The results match\n");
	}
}

void callDeviceCachedVisitBFS(Node *d_graph, int *d_size, int *d_children, int size, int *d_maxChildren, int *synchResult) {
	hipEvent_t start;
	hipEventCreate(&start);
    hipEvent_t stop;
    hipEventCreate(&stop);

    int *d_cost, *d_waveMask, *d_nextWaveMask;

	// Allocate space for device copies
	hipMalloc((void **)&d_cost, size * sizeof(int));
	hipMalloc((void **)&d_waveMask, size * sizeof(int));
	hipMalloc((void **)&d_nextWaveMask, size * sizeof(int));


    int gridSz = ceil(((float) size) / TBS);
    // Record the start event
    hipEventRecord(start, NULL);

    int *waveMask = new int[size];
    int *nextWaveMask = new int[size]; 

    int *cost = new int[size];
    cost[0] = 0;
    for (int i = 1; i < size; i++) {
    	cost[i] = -1;
    	waveMask[i] = 0;
	nextWaveMask[i] = 0;
    }

    waveMask[0] = 1;

    hipMemcpy(d_cost, cost, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_waveMask, waveMask, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_nextWaveMask, nextWaveMask, size * sizeof(int), hipMemcpyHostToDevice);
    
    bool complete = false;
    while(!complete) {

    	// Launch kernel on GPU
		exploreWave<<<gridSz, TBS>>>(d_waveMask, d_nextWaveMask, d_graph, d_children, d_cost, d_size, d_maxChildren);
		hipDeviceSynchronize();
		setPreviousExplored<<<gridSz, TBS>>>(d_waveMask, d_nextWaveMask, d_size);		
		hipDeviceSynchronize();
		hipMemcpy(d_waveMask, d_nextWaveMask, size * sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(d_nextWaveMask, nextWaveMask, size * sizeof(int), hipMemcpyHostToDevice);

		//exploreWave<<<gridSz, TBS>>>(d_waveMask, d_nextWaveMask, d_graph, d_children, d_cost, d_size, d_maxChildren);
		complete = true;
		hipMemcpy(waveMask, d_waveMask, size * sizeof(int), hipMemcpyDeviceToHost);
		for(int i = 0 ; i < size; i++){
			if(waveMask[i] == 1){
				complete = false;
			}
		}
		printf("\n");
    }

	
	
	// Make sure result is finished
	hipDeviceSynchronize();

	// Record end event
	hipEventRecord(stop, NULL);
	hipEventSynchronize(stop);
	float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);

    printf("GPU Time= %.3f msec\n", msecTotal);

	// Copy result back to host
	int *gpu_result = (int *) malloc(size * sizeof(int));
	hipMemcpy(gpu_result, d_cost, size * sizeof(int), hipMemcpyDeviceToHost);

	bool isCorrect = true;

	for (int j = 0; j < size; j++) {
		printf("%i cost: %i\n", j, gpu_result[j]);
	}


	for (int i = 0; i < size; i++) {
		if (synchResult[i] != gpu_result[i]) {
			isCorrect = false;
			printf("%i CPU: %i GPU:%i\n", i, synchResult[i], gpu_result[i]);
		}
	}

	if (!isCorrect) {
		printf("The results do not match\n");
	} else {
		printf("The results match\n");
	}
}

int main (int argc, char **argv) {
	if (argc !=3) {
		printf("\nToo few arguments!\n");
		abort();
	}

	// Get command line argument
	int size = atoi(argv[1]);
	int maxEdgesPerNode = atoi(argv[2]);

	Node* nodes = generateGraph(size);
	int* children = generateChildren(nodes, size, maxEdgesPerNode);
	int* numChildren = transformNumChildren(nodes, size);

	Node* d_graph;
	int *d_children, *d_size, *d_maxChildren, *d_numChildren;

	// Allocate space for device copies
	hipMalloc((void **)&d_graph, size * sizeof(Node));
	hipMalloc((void **)&d_size, sizeof(int));
	hipMalloc((void **)&d_maxChildren, sizeof(int));
	hipMalloc((void **)&d_children, size * maxEdgesPerNode * sizeof(int));
	hipMalloc((void **)&d_numChildren, size * sizeof(int));

	// Copy inputs to device
	hipMemcpy(d_graph, nodes, size * sizeof(Node), hipMemcpyHostToDevice);
	hipMemcpy(d_size, &size, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_maxChildren, &maxEdgesPerNode, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_children, children, size * maxEdgesPerNode * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_numChildren, numChildren, size * sizeof(int), hipMemcpyHostToDevice);

	//Synchronouse bfs
	vector< vector<Node*> > path = bfs(nodes, size);
	int *synchResult = transformBfs(path, size);

	callDeviceCachedVisitBFS(d_graph, d_size, d_children, size, d_maxChildren, synchResult);

	callChildListExploreWave(d_size, d_children, d_numChildren, size, d_maxChildren, synchResult);


	// Cleanup
	hipFree(d_graph); 
	hipFree(d_size);
	hipFree(d_children);

	return 0;
}

Node::Node(int newValue) {
	value = newValue;
	explored = 0;
}

Node::Node() {
}

__host__ __device__ int Node::getValue() {
	return value;
}

__host__ __device__ int* Node::getChildren() {
	return children;
}

__host__ __device__ int Node::getNumChildren() {
	return numChildren;
}

void Node::addChild(Node* child) {
	children[numChildren] = child->getValue();
	numChildren++;

	return;
}

void Node::printNode() {
	printf("Value: %i Children: [", value);
	for (int i = 0; i < numChildren; i++) {
		printf("%i", children[i]);
		if (i != numChildren - 1) {
			printf(", ");
		}
	}
	printf("]\n");
	return;
}

void Node::initializeChildren(int numEdges) {
	children = new int[numEdges];
}

__host__ __device__ int Node::getExplored() {
	return explored;
}

__device__ int Node::parallelSetExplored(int newExplored) {
	return atomicExch(&explored, newExplored);
}

void Node::setExplored(int newExplored) {
	explored = newExplored;
	return;
}



