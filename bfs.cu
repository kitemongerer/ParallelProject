#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <math.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Thread block size
#define TBS 512

// Warp size
#define WS 32

class Node {
private:
	int value;
	Node* children;
	int numChildren;

public:
	Node();
	Node(int);
	int getValue();
	void addChild(Node);
	Node* getChildren();
	int getNumChildren();
	void printNode();
	void initializeChildren(int);
};

/*__global__ void addBase(int *d_array, int *d_size, int *d_base) {
	int idx = blockIdx.x * TBS + threadIdx.x;
	if (idx < *d_size && idx >= TBS) {
		d_array[idx] = d_array[idx] + d_base[blockIdx.x];
	}
}

__device__ int scan_warp(int *d_array, int idx) { 
	int lane = idx % WS;

	int i = 1;
	while(i < WS && lane >= i) {
		d_array[idx] = d_array[idx - i] + d_array[idx];
		i *= 2;
	}

	return (lane > 0) ? d_array[idx - 1] : 0; 
}

//Multiplies each element of sparse matrix by the correct vector element and puts the result back in the matrix
__global__ void allPrefixSums(int *d_array, int *d_size, int *d_base) {
	int idx = blockIdx.x * TBS + threadIdx.x;
	if (idx < *d_size) {
		int warpId = (idx / WS) % (TBS / WS);
		int lane = idx % WS;
		int arrayVal = d_array[idx];
		int val = scan_warp(d_array, idx);

		__syncthreads();
		if (lane == WS - 1){
			d_array[warpId + blockIdx.x * TBS] = d_array[idx];
		}
		__syncthreads();

		// If there are more than WS ^ 2 then more than one warp is needed
		//To calculate bases
		if (warpId == 0){
			scan_warp(d_array, idx);
		}
		__syncthreads();

		if (warpId > 0){
			val = val + d_array[warpId + blockIdx.x * TBS - 1];
		}
		__syncthreads();

		// Save block base
		if ((idx % TBS) == TBS - 1) {
			d_base[blockIdx.x] = val + arrayVal;
		}

		d_array[idx] = val;
	}
}*/

Node* generateGraph(int nNodes, int maxEdgesPerNode) {
	srand((unsigned)time(0)); 
	Node* nodes = new Node[nNodes];
	for (int i = 0; i < nNodes; i++) {
		Node* tmp = new Node(i);
		nodes[i] = *tmp;
	}

	for (int i = 0; i < nNodes; i++) {
		int numEdges = rand() % maxEdgesPerNode;
		nodes[i].initializeChildren(numEdges);
		for (int j = 0; j < numEdges; j++) {
			//TODO don't repear Children #########################################################################################################
			int child = rand() % nNodes;
			nodes[i].addChild(nodes[child]);
		}
	}
	
	for (int i = 0; i < nNodes; i++) {
		nodes[i].printNode();
	}

	return nodes; 
}

int main (int argc, char **argv) {

	// Get command line argument
	/*int size = atoi(argv[1]);

	// Create the array
	int* array = new int[size];
	int* result = new int[size];

 	// Initialize random
	srand((unsigned)time(0)); 
	 
	//Generate random numbers
	for(int i = 0; i < size; i++){ 
		array[i] = (rand() % 1000) + 1;
	}*/

	srand((unsigned)time(0));
	generateGraph(10, 3);

	/*int *d_array, *d_base, *d_size;

	// Allocate space for device copies
	hipMalloc((void **)&d_array, size * sizeof(int));
	hipMalloc((void **)&d_base, ceil(((float) size) / TBS) * sizeof(int));
	hipMalloc((void **)&d_size, sizeof(int));
	

	// Copy inputs to device
	hipMemcpy(d_array, array, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_size, &size, sizeof(int), hipMemcpyHostToDevice);

	hipEvent_t start;
	hipEventCreate(&start);
    hipEvent_t stop;
    hipEventCreate(&stop);

	// MAIN COMPUTATION, SEQUENTIAL VERSION
	result[0] = 0;
	for (int i = 1; i < size; i++) {
		result[i] = result[i-1] + array[i-1];
	}

    int gridSz = ceil(((float) size) / TBS);
    // Record the start event
    hipEventRecord(start, NULL);

	// Launch sparseMatrixMul() kernel on GPU
	allPrefixSums<<<gridSz, TBS>>>(d_array, d_size, d_base);
	
	// Make sure result is finished
	hipDeviceSynchronize();
	
	calcBase(d_base, size, d_array);

	// Record end event
	hipEventRecord(stop, NULL);
	hipEventSynchronize(stop);
	float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);

    printf("GPU Time= %.3f msec\n", msecTotal);

	// Copy result back to host
	int *gpu_result = (int *) malloc(size * sizeof(int));
	hipMemcpy(gpu_result, d_array, size * sizeof(int), hipMemcpyDeviceToHost);

	bool isCorrect = true;
	for (int i = 0; i < size; i++) {
		//printf("%i GPU: %i CPU: %i arr: %i\n", i, gpu_result[i], result[i], array[i]);
		//Print the result if it is wrong
		if (result[i] != gpu_result[i]) {
			printf("%i GPU: %i CPU: %i\n", i, gpu_result[i], result[i]);
			isCorrect = false;
		}

		if(i % TBS == TBS - 1 || i % TBS == 0) {
			//printf("%i GPU: %i CPU: %i\n", i, gpu_result[i], result[i]);
		} 
	}

	if (!isCorrect) {
		printf("The results do not match\n");
	} else {
		printf("The results match\n");
	}

	// Cleanup
	hipFree(d_array); 
	hipFree(d_size); */

	return 0;
}

Node::Node(int newValue) {
	value = newValue;
}

Node::Node() {
}

int Node::getValue() {
	return value;
}

Node* Node::getChildren() {
	return children;
}

int Node::getNumChildren() {
	return numChildren;
}

void Node::addChild(Node child) {
	children[numChildren] = child;
	numChildren++;

	return;
}

void Node::printNode() {
	printf("Value: %i Children: [", value);
	for (int i = 0; i < numChildren; i++) {
		printf("child: %i is %i, ", i, children[i].getValue());
	}
	printf("]\n");
	return;
}

void Node::initializeChildren(int numEdges) {
	children = new Node[numEdges];
}




